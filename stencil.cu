
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 100
#define R 3
#define BLOCK_SIZE 512

__global__ void singlethread_stencil(int* d_in, int* d_out, int M)
{
	int tid = threadIdx.x;
	if (tid == 0){
		for (int i = 0; i < M; i++){
			for (int j = -R; j <= R; j++)
				d_out[i] += d_in[i + R + j];
		}
	}

}

__global__ void multiplethreads_stencil(int* d_in, int* d_out, int M)
{ 
     int tid = (blockIdx.x*BLOCK_SIZE)+threadIdx.x;
	 if (tid < M){
	    int result = 0;
	    for (int j = -R; j <= R; j++)
	        result += d_in[tid + R + j];
	    d_out[tid] =  result;
				           
	    }
}

__global__ void faster_stencil(int* d_in, int* d_out, int M){
	
    __shared__ int temp[BLOCK_SIZE+(2*R)];
	
    int g_id = (blockIdx.x*BLOCK_SIZE)+threadIdx.x;
	int l_id = threadIdx.x+R;
	

	if(g_id < M) {
	    temp[l_id] = d_in[g_id+R];
	    if(threadIdx.x < R){
	    temp[l_id-R] = d_in[g_id];
	    temp[l_id+BLOCK_SIZE] = d_in[g_id+BLOCK_SIZE];
	}
	__syncthreads();

	int result = 0;
        for(int j=-R; j<=R; j++)
		result+= temp[l_id+R];

	d_out[g_id] =result;
    }
}

int main()
{
	int M = N-2*R;

	int h_in[N];
	int h_out[M];

	for (int i = 0; i < N; i++)
		h_in[i] = 1;

	int* d_in;
	int* d_out;

    float ms;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	//Part 1: Memmory tranfer f rom host to device
	hipMalloc((void**) &d_in, N*sizeof(int));
	hipMalloc((void**) &d_out, M*sizeof(int));

	hipMemcpy(d_in, &h_in, N*sizeof(int), hipMemcpyHostToDevice);

	//Part 2: Excute Kernel
    
    hipEventRecord(start);
	singlethread_stencil<<<1, M>>>(d_in, d_out, M);
    //multiplethreads_stencil<<<(int) ceil(N/(double) BLOCK_SIZE), BLOCK_SIZE>>>(d_in, d_out, M);
    //faster_stencil<<<(int) ceil(N/(double) BLOCK_SIZE), BLOCK_SIZE>>>(d_in, d_out, M);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
	//Part 3: Memory tranfer from device to host
	hipMemcpy(&h_out, d_out, M*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);

	//Part 4: Check the result
	
	for (int i = 0; i < M; i++){
		if (h_out[i] != 2*R+1){
			printf("Incorrect result!\n");
			return -1;
		}
        
        else {
            printf("Correct Result\n");
            printf("Time used: %f milliseconds\n", ms);
            return -1;
    	}
    }

}
