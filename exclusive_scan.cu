
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

#define N 8

__global__ void exclusive_scan(int *d_in)
{
    //Phase 1 (Uptree)
    int s = 1;
        __shared__ int temp_in[N];
        int i = 2*s*(threadIdx.x+1)-1;
        temp_in[i] = d_in[i];

        __syncthreads();

    for(; s<=N-1; s<<=1)
    {
        if((i-s >= 0) && (i<N)) {
            //printf("s = %d, i= %d \n", s, i);
            int a = temp_in[i];
            int b = temp_in[i-s];
            __syncthreads();
            temp_in[i] = a+b;
            //printf("Write in[%d] = %d\n", i, a+b);
        }
        __syncthreads();
    }

    //Phase 2 (Downtree)
    if(threadIdx.x == 0)
        temp_in[N-1] = 0;

    for(s = s/2; s >= 1; s>>=1)
    {
        int i = 2*s*(threadIdx.x+1)-1;
        if((i-s >= 0) && (i<N)) {
            //printf("s = %d, i= %d \n", s, i);
            int r = temp_in[i];
            int l = temp_in[i-s];
            __syncthreads();
            temp_in[i] = l+r;
            temp_in[i-s] = r;
            __syncthreads();
            //printf("Write in[%d] = %d\n", i, a+b);
        }
        __syncthreads();
    }

        d_in[i] = temp_in[i];
}

int main()
{
        int h_in[N];
        int h_out[N];

        //timer
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        h_in[0] = 3;
    h_in[1] = 1;
    h_in[2] = 7;
    h_in[3] = 0;
    h_in[4] = 4;
    h_in[5] = 1;
    h_in[6] = 6;
    h_in[7] = 3;

        int *d_in;
        //int *d_out;

        hipMalloc((void**) &d_in, N*sizeof(int));
        //cudaMalloc((void**) &d_out, N*sizeof(int));
        hipMemcpy(d_in, &h_in, N*sizeof(int), hipMemcpyHostToDevice);

        hipEventRecord(start);
        //Implementing kernel call
        exclusive_scan<<<1, 4>>>(d_in);
        hipEventRecord(stop);

        hipMemcpy(&h_out, d_in, N*sizeof(int), hipMemcpyDeviceToHost);

        hipEventSynchronize(stop);
        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);

        for(int i=0; i<N; i++)
                printf("out[%d] =  %d\n", i, h_out[i]);

    hipFree(d_in);

        printf("Time used: %f milliseconds\n", ms);

        return -1;

}
