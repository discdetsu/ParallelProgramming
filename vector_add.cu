#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void parallel_vector_add(int* d_a, int* d_b, int* d_c, int* d_n)
{
	int i = (blockIdx.x*blockIdx.x)+threadIdx.x;
	if(i < *d_n)
	{
		printf(" I am thread #%d, and about to compute c[%d]. \n", i, i);
		d_c[i] = d_a[i] + d_b[i];
	}

	else
	{
		printf("I am thread #%d, and doing nothing.\n", i);
	}
}

int main()
{
	int n;
	scanf("%d", &n);
	
	// Declare input and output on host
	int h_a[n];
	int h_b[n];
	int h_c[n];
	for(int i = 0; i < n; i++)
	{
	    h_a[i] = i;
	    h_b[i] = n - i;
	}

	// PART I: Copy data from host to device
	int* d_a, *d_b, *d_c, *d_n;
	hipMalloc((void **) &d_a, n*sizeof(int));
	hipMalloc((void **) &d_b, n*sizeof(int));
	hipMalloc((void **) &d_c, n*sizeof(int));
	hipMalloc((void **) &d_n, sizeof(int));

	hipMemcpy(d_a, &h_a, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &h_b, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_n, &n, sizeof(int),  hipMemcpyHostToDevice);

	// PART II: Kernel launch
	int max=(n/512);
	parallel_vector_add<<<max,512>>>(d_a,d_b,d_c,d_n);
        hipDeviceSynchronize();
	
	// PART III: Copy data from device back to host, and free all allocate on device
	hipMemcpy(&h_c, d_c, n*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	for (int i = 0; i < n; i++){
		printf("%d", h_c[i]);
}
