
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 20

void add(int *X, int *Y, int *Z)
{
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            Z[i*N+j] = X[i*N+j] + Y[i*N+j];
}

__global__ void add_kernel(int *X, int *Y, int *Z)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	
	Z[i*N+j] = X[i*N+j] + Y[i*N+j];

}

int main()
{
    //Input matrix
    int X[N*N];
    int Y[N*N];
    
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++){
            X[i*N+j] = 0;
            Y[i*N+j] = 1;
        }
    //Output matrix
    int Z[N*N];
	
	int *d_X, *d_Y, *d_Z;
	hipMalloc((void**) &d_X, (N*N)*sizeof(int));
	hipMalloc((void**) &d_Y, (N*N)*sizeof(int));
	hipMalloc((void**) &d_Z, (N*N)*sizeof(int));

	hipMemcpy(d_X, &X, (N*N)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_Y, &Y, (N*N)*sizeof(int), hipMemcpyHostToDevice);
	dim3 dimGrid(1,1,1);
	dim3 dimBlock(20,20,1);
	add_kernel<<<dimGrid, dimBlock>>>(d_X, d_Y, d_Z);	
    //add(X, Y, Z);

	hipMemcpy(&Z, d_Z, (N*N)*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_X);
	hipFree(d_Y);
	hipFree(d_Z);
	
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++) {
			printf("%d ", Z[i*N+j]);
		}
	    printf("\n");
	}

    return -1;
}

