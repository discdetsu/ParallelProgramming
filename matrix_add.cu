
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 800

void add(int *X, int *Y, int *Z)
{
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++)
            Z[i*N+j] = X[i*N+j] + Y[i*N+j];
}

__global__ void add_kernel(int *X, int *Y, int *Z)
{
	int i = threadIdx.x;
	int j = threadIdx.y;
	
	Z[i*N+j] = X[i*N+j] + Y[i*N+j];

}

int main()
{
    //Input matrix
    int X[N*N];
    int Y[N*N];
    
    for (int i = 0; i < N; i++)
        for (int j = 0; j < N; j++){
            X[i*N+j] = 0;
            Y[i*N+j] = 1;
        }
    //Output matrix
    int Z[N*N];
	
	int *d_X, *d_Y, *d_Z;
	hipMalloc((void**) &d_X, (N*N)*sizeof(int));
	hipMalloc((void**) &d_Y, (N*N)*sizeof(int));
	hipMalloc((void**) &d_Z, (N*N)*sizeof(int));
	
	//timer
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpy(d_X, &X, (N*N)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_Y, &Y, (N*N)*sizeof(int), hipMemcpyHostToDevice);
	dim3 dimGrid(32,1,1);
	dim3 dimBlock(32,1,1);
	
	hipEventRecord(start);
	add_kernel<<<dimGrid, dimBlock>>>(d_X, d_Y, d_Z);
	hipEventRecord(stop);	
    //add(X, Y, Z);

	hipMemcpy(&Z, d_Z, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float ms = 0;
	hipEventElapsedTime(&ms, start, stop);

	hipFree(d_X);
	hipFree(d_Y);
	hipFree(d_Z);
	
	int sum = 0;	
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++) {
			sum += Z[i*N+j];
		}
	}
	
	printf("Time used: %f milliseconds\n", ms);
    return -1;
}
